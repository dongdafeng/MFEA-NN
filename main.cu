#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <fstream>
#include <cstdint>
#include <typeinfo>
#include <vector>
#include <thread>
#include <algorithm>
#include <cmath>
#include <limits>
#include <array>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hipDNN.h>

#include "utils.h"
#include "Matrix.hpp"


//#include "MnistHelper.hpp"
//#include "DatasetIonosphereHelper.hpp"
//#include "DatasetTictactoeHelper.hpp"
#include "NbitHelper.hpp"



#include "MFEAChromosome.hpp"
#include "MFEA.hpp"
#include "MFEATask.hpp"


// variables for training data
DATATYPE* training_input_data;
DATATYPE* training_output_data;

// variables for testing data
DATATYPE* testing_input_data;
DATATYPE* testing_output_data;


void showMUltitasksSetting() {
	std::cout << "Multitasks setting:" << std::endl;;
    for (uint32_t task = 0; task < TASK_SIZE; ++task) {
		std::cout << "--- Task " << task << " has " << getNumberofLayersbyTask(task) << " layers (including " << getNumberofLayersbyTask(task) - 1 << " hidden layers)" << std::endl;
		for (uint32_t layer = 1; layer < getNumberofLayersbyTask(task) + 1; ++layer) {
			if (layer == getNumberofLayersbyTask(task)) {
				std::cout << "----- Layer " << layer << " (output layer): " << getNumberofUnitsbyTaskLayer(task, layer) << " units" << std::endl;
			} else {
				std::cout << "----- Layer " << layer << ": " << getNumberofUnitsbyTaskLayer(task, layer) << " units" << std::endl;
			}
			std::cout << "------- Data offset    = " << std::get<OFFSET_IDX>(getLayerWeightsandBiasesbyTaskLayer(task, layer))
							 << "\t Data size    = " << std::get<SIZE_IDX>(getLayerWeightsandBiasesbyTaskLayer(task, layer)) << std::endl;
			std::cout << "------- Weights offset = " << std::get<OFFSET_IDX>(getLayerWeightsbyTaskLayer(task, layer))
							 << "\t Weights size = " << std::get<SIZE_IDX>(getLayerWeightsbyTaskLayer(task, layer)) << std::endl;
			std::cout << "------- Biases offset  = " << std::get<OFFSET_IDX>(getLayerBiasesbyTaskLayer(task, layer))
							 << "\t Biases size  = " << std::get<SIZE_IDX>(getLayerBiasesbyTaskLayer(task, layer)) << std::endl;
		}
	}
}

void testDecode();
void testSBX();
void testPMU();
void testEval();


int main(int argc, char** argv) {
	/* manually set device for running */
	int device_id;
	if (argc > 1) {
		device_id = atoi(argv[1]);
		
	} else {
		device_id = 0;
	}
	hipSetDevice(device_id);

	
	/* load input data */
	loadDataFile<DATATYPE>(training_input_data, training_output_data, testing_input_data, testing_output_data);


	/* Total CPU Page faults: 1384 for float */
	/* Total CPU Page faults: 2477 for double */
	
	{// limit scope for object destruct before destroy CUDA environment
		MFEA<90, 1000, 2> mfea(training_input_data, training_output_data,
							testing_input_data, testing_output_data,
							device_id);
		if (mfea.init_libraries() != 0) {
			return EXIT_FAILURE;
		}
		
		mfea.initialize();	// does not cause page fault
		mfea.evolution();	// does not cause page fault
		mfea.sumariseResults();
		mfea.writeSumaryResults();
		mfea.reEvaluateTheFinalPopulation();
		
		
		/*for (uint32_t i = 0; i < 200; ++i) {
			float __cf_distributionindex		= 1.0 * (std::rand() % 11);			// randomize between 0 - 10
			float __mf_randommatingprobability	= 1.0;
			float __mf_polynomialmutationindex	= 1.0 * (std::rand() % 11);			// randomize between 0 - 10
			float __mf_mutationratio			= 0.05 * (1 + std::rand() % 10);	// randomize between 5% - 50%
			mfea.setTunableFactors(__cf_distributionindex,
										__mf_randommatingprobability,
										__mf_polynomialmutationindex,
										__mf_mutationratio	);
			
			mfea.initialize();
			mfea.evolution();
			mfea.sumariseResults();
			mfea.writeSumaryResults();
		}*/
		

		mfea.finalize_libraries();
	}
	
	showMUltitasksSetting();

	
    /* Reset CUDA evironment */
    hipDeviceReset();
    
	return 0;
}

/*int main(int argc, char** argv) {
	// manually set device for running
	int device_id;
	if (argc > 1) {
		device_id = atoi(argv[1]);
		
	} else {
		device_id = 0;
	}
	hipSetDevice(device_id);

	hipblasHandle_t cublas_handle;
	cublasCALL(hipblasCreate(&cublas_handle));
	
	hipdnnHandle_t cudnn_handle;
	cudnnCALL(hipdnnCreate(&cudnn_handle));

	hiprandGenerator_t curand_prng;
	// Create a pseudo-random number generator
	hiprandCreateGenerator(&curand_prng, HIPRAND_RNG_PSEUDO_MTGP32);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(curand_prng, 0);

	
	// load input data
	loadDataFile<DATATYPE>(training_input_data, training_output_data, testing_input_data, testing_output_data);



	testEval();

	
	
	showMUltitasksSetting();

	
    // Reset CUDA evironment
    hipDeviceReset();
    
	return 0;
}*/

void testDecode() {
	hipblasHandle_t cublas_handle;
	cublasCALL(hipblasCreate(&cublas_handle));
	
	hipdnnHandle_t cudnn_handle;
	cudnnCALL(hipdnnCreate(&cudnn_handle));

	hiprandGenerator_t curand_prng;
	// Create a pseudo-random number generator
	hiprandCreateGenerator(&curand_prng, HIPRAND_RNG_PSEUDO_MTGP32);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(curand_prng, 0);

	std::array<MFEA_Chromosome, 4> population;
	thrust::for_each(population.begin(), population.end(), MFEA_Chromosome_Randomize(curand_prng));

	hipDeviceSynchronize();
	std::cout << population[0];


	for (uint32_t i = 0; i < getTotalLayerWeightsandBiases(); ++i) {
		population[0].rnvec[i] = i;
	}
	printMatrix<DATATYPE>(1, getTotalLayerWeightsandBiases(), population[0].rnvec);
	//cublas_transposeMatrix<DATATYPE>(6, 8, population[0].rnvec, population[1].rnvec, cublas_handle);

	DATATYPE* W;
	CUDA_M_MALLOC_MANAGED(W, DATATYPE, getTotalLayerWeightsandBiases());

	for (uint32_t task = 0; task < TASK_SIZE; ++task) {
		for (uint32_t layer = 1; layer <= getNumberofLayersbyTask(task); ++layer) {
			std::tuple<uint32_t, uint32_t> shape = population[0].decode(population[0].rnvec, W, task, layer, cublas_handle);
			hipDeviceSynchronize();
			std::cout << "W for task " << task << " layer " << layer << " : " << std::endl;
			printMatrix<DATATYPE>(std::get<MATRIX_NROW>(shape), std::get<MATRIX_NCOL>(shape), W);


			std::tuple<uint32_t, uint32_t> bias = getLayerBiasesbyTaskLayer(task, layer);
			std::cout << "b for task " << task << " layer " << layer << " : " << std::endl;
			printMatrix<DATATYPE>(1, std::get<SIZE_IDX>(bias), population[0].rnvec + std::get<OFFSET_IDX>(bias));
		}
	}
}

void testSBX() {
	hiprandGenerator_t curand_prng;
	// Create a pseudo-random number generator
	hiprandCreateGenerator(&curand_prng, HIPRAND_RNG_PSEUDO_MTGP32);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(curand_prng, 0);
	
	std::array<MFEA_Chromosome, 4> population;
	thrust::for_each(population.begin(), population.end(), MFEA_Chromosome_Randomize(curand_prng));


	DATATYPE* ct_beta;
	cudaCALL(CUDA_M_MALLOC_MANAGED(ct_beta, DATATYPE, getTotalLayerWeightsandBiases()));



	hipDeviceSynchronize();
	BUG(getTotalLayerWeightsandBiases());
	for (uint32_t i = 0; i < getTotalLayerWeightsandBiases(); ++i) {
		population[0].rnvec[i] = double(i) / getTotalLayerWeightsandBiases();
		population[1].rnvec[i] = double(i) / getTotalLayerWeightsandBiases();
	}

	test_crossover(population[0], population[1],
						  population[2], population[3],
						  5, ct_beta,
						  curand_prng);

	examineCrossover(population[0], population[1], population[2], population[3]);

	hipDeviceSynchronize();
	
	std::cout << population[0];
	std::cout << population[1];
	std::cout << population[2];
	std::cout << population[3];
}

void testPMU() {
	hiprandGenerator_t curand_prng;
	// Create a pseudo-random number generator
	hiprandCreateGenerator(&curand_prng, HIPRAND_RNG_PSEUDO_MTGP32);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(curand_prng, 0);
	
	std::array<MFEA_Chromosome, 4> population;
	thrust::for_each(population.begin(), population.end(), MFEA_Chromosome_Randomize(curand_prng));


	DATATYPE* ct_beta;
	DATATYPE* rp;
	cudaCALL(CUDA_M_MALLOC_MANAGED(ct_beta, DATATYPE, getTotalLayerWeightsandBiases()));
	cudaCALL(CUDA_M_MALLOC_MANAGED(rp, DATATYPE, getTotalLayerWeightsandBiases()));


	hipDeviceSynchronize();
	BUG(getTotalLayerWeightsandBiases());
	for (uint32_t i = 0; i < getTotalLayerWeightsandBiases(); ++i) {
		population[0].rnvec[i] = double(i) / getTotalLayerWeightsandBiases();
	}

	test_mutate(population[0], population[1],
						5, 1,
						ct_beta, rp, curand_prng);

	hipDeviceSynchronize();
	std::cout << population[1];
}

void testEval() {
	hipblasHandle_t cublas_handle;
	cublasCALL(hipblasCreate(&cublas_handle));
	
	hipdnnHandle_t cudnn_handle;
	cudnnCALL(hipdnnCreate(&cudnn_handle));

	hiprandGenerator_t curand_prng;
	// Create a pseudo-random number generator
	hiprandCreateGenerator(&curand_prng, HIPRAND_RNG_PSEUDO_MTGP32);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(curand_prng, 0);
	
	std::array<MFEA_Chromosome, 4> population;
	thrust::for_each(population.begin(), population.end(), MFEA_Chromosome_Randomize(curand_prng));



	DATATYPE* dev_mat_temp_rnvec;
	DATATYPE* dev_mat_temp_w;
	DATATYPE* dev_mat_ones;
	std::array<DATATYPE*, LAYER_SIZE + 1> dev_mat_temp_layers;
	cudaCALL(CUDA_M_MALLOC_MANAGED(dev_mat_temp_rnvec, DATATYPE, getTotalLayerWeightsandBiases()));
	cudaCALL(CUDA_M_MALLOC_MANAGED(dev_mat_temp_w, DATATYPE, getTotalLayerWeightsandBiases()));
	cudaCALL(CUDA_M_MALLOC_MANAGED(dev_mat_ones, DATATYPE, TRAINING_SIZE));
	cuda_fillMatrix<DATATYPE>(TRAINING_SIZE, 1, dev_mat_ones, 1.0f);
	
	for (uint32_t i = 0; i < LAYER_SIZE + 1; ++i) {
		cudaCALL(CUDA_M_MALLOC_MANAGED(dev_mat_temp_layers[i], DATATYPE, TRAINING_SIZE * getNumberofUnitsbyTaskLayer(TASKINDEX_LARGEST, i)));
	}


	hipDeviceSynchronize();
	BUG(getTotalLayerWeightsandBiases());
	for (uint32_t i = 0; i < getTotalLayerWeightsandBiases(); ++i) {
		population[0].rnvec[i] = double(i) / getTotalLayerWeightsandBiases();
	}

	printGPUArray(dev_mat_ones, TRAINING_SIZE);
	printGPUArray(dev_mat_temp_rnvec, getTotalLayerWeightsandBiases());


	int i = 0;
	//for (uint32_t i = 0; i < 1; ++i) {
		population[i].skill_factor = i % TASK_SIZE;
		population[i].evalObj(TRAINING_SIZE, OUTPUT_SIZE,
								training_input_data,
								training_output_data,
								dev_mat_temp_rnvec,
								dev_mat_temp_w,
								dev_mat_ones,
								dev_mat_temp_layers,
								cublas_handle, cudnn_handle,
								true);
	//}

	printGPUArray(dev_mat_temp_layers[LAYER_SIZE - 1], TRAINING_SIZE * getNumberofUnitsbyTaskLayer(TASKINDEX_LARGEST, LAYER_SIZE));

	hipDeviceSynchronize();
	std::cout << population[0];
}